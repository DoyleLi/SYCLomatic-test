// ====------ cublas_64.cu ------------------------------- *- CUDA -* ---==== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ====------------------------------------------------------------------==== //


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const half *alpha_h;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  const half *A_h;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  const half *B_h;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  const half *beta_h;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  half *C_h;
  float *C1_s;
  double *C1_d;
  float2 *C1_c;
  double2 *C1_z;
  int64_t ldc;
  hipblasFillMode_t uplo;
  hipblasSideMode_t side;
  hipblasDiagType_t diag;
  int64_t result;
  float result_s;
  double result_d;
  float2 result_c;
  double2 result_z;
  int64_t incx;
  int64_t incy;

  int64_t elemSize;
  hipStream_t stream;
  status = cublasSetVector_64(n, elemSize, A_s, incx, C_s, incy);
  status = cublasGetVector_64(n, elemSize, A_s, incx, C_s, incy);
  status = cublasSetVectorAsync_64(n, elemSize, A_s, incx, C_s, incy, stream);
  status = cublasGetVectorAsync_64(n, elemSize, A_s, incx, C_s, incy, stream);

  status = cublasSetMatrix_64(m, n, elemSize, A_s, lda, C_s, ldb);
  status = cublasGetMatrix_64(m, n, elemSize, A_s, lda, C_s, ldb);
  status = cublasSetMatrixAsync_64(m, n, elemSize, A_s, lda, C_s, ldb, stream);
  status = cublasGetMatrixAsync_64(m, n, elemSize, A_s, lda, C_s, ldb, stream);

  status = hipblasIsamax_64(handle, n, A_s, lda, &result);
  status = hipblasIdamax_64(handle, n, A_d, lda, &result);
  status = hipblasIcamax_64(handle, n, A_c, lda, &result);
  status = hipblasIzamax_64(handle, n, A_z, lda, &result);

  status = hipblasIsamin_64(handle, n, A_s, lda, &result);
  status = hipblasIdamin_64(handle, n, A_d, lda, &result);
  status = hipblasIcamin_64(handle, n, A_c, lda, &result);
  status = hipblasIzamin_64(handle, n, A_z, lda, &result);

  status = hipblasSnrm2_64(handle, n, A_s, incx, &result_s);
  status = hipblasDnrm2_64(handle, n, A_d, incx, &result_d);
  status = hipblasScnrm2_64(handle, n, A_c, incx, &result_s);
  status = hipblasDznrm2_64(handle, n, A_z, incx, &result_d);

  status = hipblasSdot_64(handle, n, A_s, incx, B_s, incy, &result_s);
  status = hipblasDdot_64(handle, n, A_d, incx, B_d, incy, &result_d);
  status = hipblasCdotu_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasCdotc_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasZdotu_64(handle, n, A_z, incx, B_z, incy, &result_z);
  status = hipblasZdotc_64(handle, n, A_z, incx, B_z, incy, &result_z);

  status = hipblasSscal_64(handle, n, alpha_s, C_s, incx);
  status = hipblasDscal_64(handle, n, alpha_d, C_d, incx);
  status = hipblasCscal_64(handle, n, alpha_c, C_c, incx);
  status = hipblasZscal_64(handle, n, alpha_z, C_z, incx);
  status = hipblasCsscal_64(handle, n, alpha_s, C_c, incx);
  status = hipblasZdscal_64(handle, n, alpha_d, C_z, incx);

  status = hipblasSaxpy_64(handle, n, alpha_s, A_s, incx, C_s, incy);
  status = hipblasDaxpy_64(handle, n, alpha_d, A_d, incx, C_d, incy);
  status = hipblasCaxpy_64(handle, n, alpha_c, A_c, incx, C_c, incy);
  status = hipblasZaxpy_64(handle, n, alpha_z, A_z, incx, C_z, incy);

  status = hipblasScopy_64(handle, n, A_s, incx, C_s, incy);
  status = hipblasDcopy_64(handle, n, A_d, incx, C_d, incy);
  status = hipblasCcopy_64(handle, n, A_c, incx, C_c, incy);
  status = hipblasZcopy_64(handle, n, A_z, incx, C_z, incy);

  status = hipblasSswap_64(handle, n, C_s, incx, C1_s, incy);
  status = hipblasDswap_64(handle, n, C_d, incx, C1_d, incy);
  status = hipblasCswap_64(handle, n, C_c, incx, C1_c, incy);
  status = hipblasZswap_64(handle, n, C_z, incx, C1_z, incy);

  status = hipblasSasum_64(handle, n, A_s, incx, &result_s);
  status = hipblasDasum_64(handle, n, A_d, incx, &result_d);
  status = hipblasScasum_64(handle, n, A_c, incx, &result_s);
  status = hipblasDzasum_64(handle, n, A_z, incx, &result_d);

  const float *const_s;
  const double *const_d;
  const float2 *const_c;
  const double2 *const_z;
  float *s;
  double *d;
  float2 *c;
  double2 *z;

  status = hipblasSrot_64(handle, n, C_s, incx, C1_s, incy, const_s, const_s);
  status = hipblasDrot_64(handle, n, C_d, incx, C1_d, incy, const_d, const_d);
  status = hipblasCrot_64(handle, n, C_c, incx, C1_c, incy, const_s, const_c);
  status = hipblasCsrot_64(handle, n, C_c, incx, C1_c, incy, const_s, const_s);
  status = hipblasZrot_64(handle, n, C_z, incx, C1_z, incy, const_d, const_z);
  status = hipblasZdrot_64(handle, n, C_z, incx, C1_z, incy, const_d, const_d);

  status = hipblasSrotm_64(handle, n, s, incx, s, incy, const_s);
  status = hipblasDrotm_64(handle, n, d, incx, d, incy, const_d);

  const float *x_s;
  const double *x_d;
  const float2 *x_c;
  const double2 *x_z;
  float *y_s;
  double *y_d;
  float2 *y_c;
  double2 *y_z;
  status = hipblasSgemv_64(handle, transa, m, n, alpha_s, A_s, lda, x_s, incx,
                          beta_s, y_s, incy);
  status = hipblasDgemv_64(handle, transa, m, n, alpha_d, A_d, lda, x_d, incx,
                          beta_d, y_d, incy);
  status = hipblasCgemv_64(handle, transa, m, n, alpha_c, A_c, lda, x_c, incx,
                          beta_c, y_c, incy);
  status = hipblasZgemv_64(handle, transa, m, n, alpha_z, A_z, lda, x_z, incx,
                          beta_z, y_z, incy);

  int64_t kl, ku;
  status = hipblasSgbmv_64(handle, transa, m, n, kl, ku, alpha_s, A_s, lda, x_s,
                          incx, beta_s, y_s, incy);
  status = hipblasDgbmv_64(handle, transa, m, n, kl, ku, alpha_d, A_d, lda, x_d,
                          incx, beta_d, y_d, incy);
  status = hipblasCgbmv_64(handle, transa, m, n, kl, ku, alpha_c, A_c, lda, x_c,
                          incx, beta_c, y_c, incy);
  status = hipblasZgbmv_64(handle, transa, m, n, kl, ku, alpha_z, A_z, lda, x_z,
                          incx, beta_z, y_z, incy);

  status = hipblasStrmv_64(handle, uplo, transa, diag, n, A_s, lda, y_s, incy);
  status = hipblasDtrmv_64(handle, uplo, transa, diag, n, A_d, lda, y_d, incy);
  status = hipblasCtrmv_64(handle, uplo, transa, diag, n, A_c, lda, y_c, incy);
  status = hipblasZtrmv_64(handle, uplo, transa, diag, n, A_z, lda, y_z, incy);

  status =
      hipblasStbmv_64(handle, uplo, transa, diag, n, k, A_s, lda, y_s, incy);
  status =
      hipblasDtbmv_64(handle, uplo, transa, diag, n, k, A_d, lda, y_d, incy);
  status =
      hipblasCtbmv_64(handle, uplo, transa, diag, n, k, A_c, lda, y_c, incy);
  status =
      hipblasZtbmv_64(handle, uplo, transa, diag, n, k, A_z, lda, y_z, incy);

  status = hipblasStpmv_64(handle, uplo, transa, diag, n, A_s, y_s, incy);
  status = hipblasDtpmv_64(handle, uplo, transa, diag, n, A_d, y_d, incy);
  status = hipblasCtpmv_64(handle, uplo, transa, diag, n, A_c, y_c, incy);
  status = hipblasZtpmv_64(handle, uplo, transa, diag, n, A_z, y_z, incy);

  status = hipblasStrsv_64(handle, uplo, transa, diag, n, A_s, lda, y_s, incy);
  status = hipblasDtrsv_64(handle, uplo, transa, diag, n, A_d, lda, y_d, incy);
  status = hipblasCtrsv_64(handle, uplo, transa, diag, n, A_c, lda, y_c, incy);
  status = hipblasZtrsv_64(handle, uplo, transa, diag, n, A_z, lda, y_z, incy);

  status = hipblasStpsv_64(handle, uplo, transa, diag, n, A_s, y_s, incy);
  status = hipblasDtpsv_64(handle, uplo, transa, diag, n, A_d, y_d, incy);
  status = hipblasCtpsv_64(handle, uplo, transa, diag, n, A_c, y_c, incy);
  status = hipblasZtpsv_64(handle, uplo, transa, diag, n, A_z, y_z, incy);

  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda,
                          B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda,
                          B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda,
                          B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda,
                          B_z, ldb, beta_z, C_z, ldc);
  status = hipblasHgemm_64(handle, transa, transb, m, n, k, alpha_h, A_h, lda,
                          B_h, ldb, beta_h, C_h, ldc);

  status = cublasCgemm3m_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda,
                            B_c, ldb, beta_c, C_c, ldc);
  status = cublasZgemm3m_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda,
                            B_z, ldb, beta_z, C_z, ldc);

  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, beta_s,
                          C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, beta_d,
                          C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, beta_c,
                          C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, beta_z,
                          C_z, ldc);

  status = hipblasSsymm_64(handle, side, uplo, m, n, alpha_s, A_s, lda, B_s, ldb,
                          beta_s, C_s, ldc);
  status = hipblasDsymm_64(handle, side, uplo, m, n, alpha_d, A_d, lda, B_d, ldb,
                          beta_d, C_d, ldc);
  status = hipblasCsymm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb,
                          beta_c, C_c, ldc);
  status = hipblasZsymm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb,
                          beta_z, C_z, ldc);

  status = hipblasStrsm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s,
                          lda, C_s, ldc);
  status = hipblasDtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d,
                          lda, C_d, ldc);
  status = hipblasCtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c,
                          lda, C_c, ldc);
  status = hipblasZtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z,
                          lda, C_z, ldc);

  status = hipblasChemm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb,
                          beta_c, C_c, ldc);
  status = hipblasZhemm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb,
                          beta_z, C_z, ldc);

  status = hipblasCherk_64(handle, uplo, transa, n, k, alpha_s, A_c, lda, beta_s,
                          C_c, ldc);
  status = hipblasZherk_64(handle, uplo, transa, n, k, alpha_d, A_z, lda, beta_d,
                          C_z, ldc);

  status = hipblasSsyr2k_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s,
                           ldb, beta_s, C_s, ldc);
  status = hipblasDsyr2k_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d,
                           ldb, beta_d, C_d, ldc);
  status = hipblasCsyr2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c,
                           ldb, beta_c, C_c, ldc);
  status = hipblasZsyr2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z,
                           ldb, beta_z, C_z, ldc);

  status = hipblasCher2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c,
                           ldb, beta_s, C_c, ldc);
  status = hipblasZher2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z,
                           ldb, beta_d, C_z, ldc);

  status = hipblasSgeam_64(handle, transa, transb, m, n, alpha_s, A_s, lda,
                          beta_s, B_s, ldb, C_s, ldc);
  status = hipblasDgeam_64(handle, transa, transb, m, n, alpha_d, A_d, lda,
                          beta_d, B_d, ldb, C_d, ldc);
  status = hipblasCgeam_64(handle, transa, transb, m, n, alpha_c, A_c, lda,
                          beta_c, B_c, ldb, C_c, ldc);
  status = hipblasZgeam_64(handle, transa, transb, m, n, alpha_z, A_z, lda,
                          beta_z, B_z, ldb, C_z, ldc);

  status = hipblasSdgmm_64(handle, side, m, n, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDdgmm_64(handle, side, m, n, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCdgmm_64(handle, side, m, n, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZdgmm_64(handle, side, m, n, A_z, lda, B_z, ldb, C_z, ldc);

  status = hipblasStrmm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s,
                          lda, B_s, ldb, C_s, ldc);
  status = hipblasDtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d,
                          lda, B_d, ldb, C_d, ldc);
  status = hipblasCtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c,
                          lda, B_c, ldb, C_c, ldc);
  status = hipblasZtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z,
                          lda, B_z, ldb, C_z, ldc);

  status = hipblasSsyrkx_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s,
                           ldb, beta_s, C_s, ldc);
  status = hipblasDsyrkx_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d,
                           ldb, beta_d, C_d, ldc);
  status = hipblasCsyrkx_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c,
                           ldb, beta_c, C_c, ldc);
  status = hipblasZsyrkx_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z,
                           ldb, beta_z, C_z, ldc);

  status = hipblasCherkx_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c,
                           ldb, beta_s, C_c, ldc);
  status = hipblasZherkx_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z,
                           ldb, beta_d, C_z, ldc);
}
